
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void update_device_data_kernel( unsigned long size, unsigned long order, unsigned long* A, unsigned long* B, unsigned long* C, double* residual, double* x )
{
    unsigned long const index = blockDim.x * blockIdx.x + threadIdx.x;
    if ( index >= size ) return;

    unsigned long* a = A + index * 4;
    unsigned long* b = B + index * 4;
    unsigned long* c = C + index * 4;
    double* alpha = x;
    double* beta = x + 4 * order;
    double* gamma = beta + 4 * order;

    double const weigh = c[0] + c[1] + c[2] + c[3] + 1.0;

#if 0
    printf( "size = %ld\n", size );
    printf( "order = %ld\n", order );
    printf( "index = %ld\n", index );
    printf( "weigh = %lf\n", weigh );

    printf( "a[0] = %ld\n", a[0] );
    printf( "a[1] = %ld\n", a[1] );
    printf( "a[2] = %ld\n", a[2] );
    printf( "a[3] = %ld\n", a[3] );

    printf( "b[0] = %ld\n", b[0] );
    printf( "b[1] = %ld\n", b[1] );
    printf( "b[2] = %ld\n", b[2] );
    printf( "b[3] = %ld\n", b[3] );

    printf( "c[0] = %ld\n", c[0] );
    printf( "c[1] = %ld\n", c[1] );
    printf( "c[2] = %ld\n", c[2] );
    printf( "c[3] = %ld\n", c[3] );
#endif
    double ss[6];
    double cc[4];

    for ( unsigned long jndex = 0; jndex < order; ++jndex )
    {
        unsigned long const offset = jndex * 4;
        double* alpha_ = alpha + offset;
        double* beta_ = beta + offset;
        ss[jndex] = 1.0*a[0]*alpha_[0] + a[1]*alpha_[1] + a[2]*alpha_[2] + a[3]*alpha_[3];
        ss[jndex] *= 1.0*b[0]*beta_[0] + b[1]*beta_[1] + b[2]*beta_[2] + b[3]*beta_[3];
#if 0
        printf( "ss[%ld/%ld] = %lf\n", jndex, order, ss[jndex] );
#endif
    }


    cc[0] = -1.0 * c[0]; cc[1] = -1.0 * c[1]; cc[2] = -1.0 * c[2]; cc[3] = -1.0 * c[3];

#if 0
    printf( "cc[0] = %lf\n", cc[0] );
    printf( "cc[1] = %lf\n", cc[1] );
    printf( "cc[2] = %lf\n", cc[2] );
    printf( "cc[3] = %lf\n", cc[3] );
    printf( "\n" );
#endif

    for ( unsigned long kndex = 0; kndex != order; ++kndex )
    {
#if 0
        double cc0 = gamma[kndex] * ss[kndex];
        double cc1 = gamma[kndex+order] * ss[kndex];
        double cc2 = gamma[kndex+order+order] * ss[kndex];
        double cc3 = gamma[kndex+order+order+order] * ss[kndex];
        printf( "kndex %ld\n", kndex );
        printf( "cc0 = %lf\n", cc0 );
        printf( "cc1 = %lf\n", cc1 );
        printf( "cc2 = %lf\n", cc2 );
        printf( "cc3 = %lf\n", cc3 );
        printf( "\n" );
#endif
        cc[0] += gamma[kndex] * ss[kndex];
        cc[1] += gamma[kndex+order] * ss[kndex];
        cc[2] += gamma[kndex+order+order] * ss[kndex];
        cc[3] += gamma[kndex+order+order+order] * ss[kndex];
    }
#if 0
    printf( "cc[0] = %lf\n", cc[0] );
    printf( "cc[1] = %lf\n", cc[1] );
    printf( "cc[2] = %lf\n", cc[2] );
    printf( "cc[3] = %lf\n", cc[3] );
#endif

    residual[index] = ( cc[0] + cc[1] + cc[2] + cc[3] ) / weigh;

}

void update_device_data( unsigned long size, unsigned long order, unsigned long* A, unsigned long* B, unsigned long* C, double* residual, double* x )
{
    update_device_data_kernel<<<1, size>>>( size, order, A, B, C, residual, x );
    hipDeviceSynchronize();
}
