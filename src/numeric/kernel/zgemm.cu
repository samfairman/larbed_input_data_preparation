#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

//hipDoubleComplex

//C = alpha * A * B + beta * c

__device__ __inline__ hipDoubleComplex 
dc_multiply( hipDoubleComplex x, hipDoubleComplex y )
{
    double const a = x.x;
    double const b = x.x;
    double const c = y.x;
    double const d = y.y;
    //double const ac = a * c;
    double const ac = __dmul_rz( a, c );
    //double const bd = b * d;
    double const bd = __dmul_rz( b, d );
    //double const ab_cd = ( a + b ) * ( c + d );
    double const ab_cd = __dmul_rz ( __dadd_rz( a, b ) * __dadd_rz( c, d ) );

    //double real = ac - bd;
    double const real = __dsub_rz( ac, bd );
    //double imag = ab_cd - ac - bd;
    //double const imag = __dsub_rz( ab_cd, __dmul_rz( ac, bd ) );
    double const imag = __dsub_rz( ab_cd, __dmul_rz( ac, bd ) );
    return make_hipDoubleComplex( real, imag );
}

__global__ void
zgemm( const hipDoubleComplex* __restrict__ A, const hipDoubleComplex* __restrict__ B, hipDoubleComplex* __restrict__ C, const hipDoubleComplex* __restrict__ alpha, const hipComplex* __restrict__ beta, int const dim )
{
    hipDoubleComplex CValue = make_hipDoubleComplex( 0.0, 0.0 );

    int const bidx = blockIdx.x;
    int const bidy = blockIdx.y;
    int const tidx = threadIdx.x;
    int const tidy = threadIdx.y;

    int const row = ( bidx << 4 ) + tidx;
    int const col = ( bidy << 4 ) + tidy;

    __shared__ hipDoubleComplex As[16][17];
    __shared__ hipDoubleComplex Bs[16][17];

    int const aligned_blocks = dim >> 4;
    int const rest_elems = dim - aligned_blocks << 4;

    for ( int block_index = 0; block_index != aligned_blocks; ++block_index )
    {

    }

    for ( int k = 0; k < ( 16 + dim - 1 ) / 16; k++ )
    {
        if ( k * 16 + threadIdx.x < dim && row < dim )
            As[threadIdx.y][threadIdx.x] = A[row * dim + k * 16 + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0;

        if ( k * 16 + threadIdx.y < Brows && col < dim )
            Bs[threadIdx.y][threadIdx.x] = B[( k * 16 + threadIdx.y ) * dim + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for ( int n = 0; n < 16; ++n )
            //CValue += cuCml( As[threadIdx.y][n], Bs[n][threadIdx.x] );
            CValue += dc_multiply( As[threadIdx.y][n], Bs[n][threadIdx.x] );

        __syncthreads();
    }

    if ( row < dim && col < dim )
        C[( ( blockIdx.y * blockDim.y + threadIdx.y )*dim ) + ( blockIdx.x * blockDim.x ) + threadIdx.x] = CValue;

}
