#include <hip/hip_runtime.h>

void __global__
cm_eye_impl( float2* v, unsigned long n )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;
    
    if ( tx < n )
        v[ tx * (n+1) ].x = 1.0;
}

void cm_eye( float2* v, unsigned long n )
{
    const unsigned long grids = ( n + 1023 ) >> 10;
    cm_eye_impl<<<grids, 1024>>>( v, n );    
}

void __global__
d_cm_eye_impl( double2* v, unsigned long n )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;
    
    if ( tx < n )
        v[ tx * (n+1) ].x = 1.0;
}

void cm_eye( double2* v, unsigned long n )
{
    const unsigned long grids = ( n + 1023 ) >> 10;
    d_cm_eye_impl<<<grids, 1024>>>( v, n );    
}
