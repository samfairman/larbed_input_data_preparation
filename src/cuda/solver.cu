#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <cusp/array1d.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

#include <cassert>
#include <iostream>


void status_check( hipsolverStatus_t status )
{
    if (status == HIPSOLVER_STATUS_NOT_INITIALIZED)
    {
        assert( !"The cuSolver library was not initialized. This is usually caused by the lack of a prior call, an error in the CUDA Runtime API called by the cuSolver routine, or an error in the hardware setup. To correct: call cusolverCreate() prior to the function call; and check that the hardware, an appropriate version of the driver, and the cuSolver library are correctly installed." );
    }

    if (status == HIPSOLVER_STATUS_ALLOC_FAILED )
        assert( !"Resource allocation failed inside the cuSolver library. This is usually caused by a hipMalloc() failure. To correct: prior to the function call, deallocate previously allocated memory as much as possible. " );

    if (status == HIPSOLVER_STATUS_INVALID_VALUE)
        assert( !"An unsupported value or parameter was passed to the function (a negative vector size, for example). To correct: ensure that all the parameters being passed have valid values. " );

    if (status == HIPSOLVER_STATUS_ARCH_MISMATCH )
        assert( !"The function requires a feature absent from the device architecture; usually caused by the lack of support for atomic operations or double precision. To correct: compile and run the application on a device with compute capability 2.0 or above. " );

    if (status == HIPSOLVER_STATUS_EXECUTION_FAILED )
        assert( !"The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons. To correct: check that the hardware, an appropriate version of the driver, and the cuSolver library are correctly installed. " );

    if (status == HIPSOLVER_STATUS_INTERNAL_ERROR )
        assert( !"An internal cuSolver operation failed. This error is usually caused by a hipMemcpyAsync() failure.  To correct: check that the hardware, an appropriate version of the driver, and the cuSolver library are correctly installed. Also, check that the memory passed as a parameter to the routine is not being deallocated prior to the routine’s completion. " );

    if (status == HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED )
        assert( !"The matrix type is not supported by this function. This is usually caused by passing an invalid matrix descriptor to the function. To correct: check that the fields in descrA were set correctly. " );
}


// link -lcusparse_static   -lculibos

float solve( int row, int col, int size, int* A_row, int* A_col, float* A_data, float* x, float* b )
{
    //copy everything to device
    thrust::device_vector<int> DA_row( A_row, A_row+row );
    thrust::device_vector<int> DA_col( A_col, A_col+col );
    thrust::device_vector<float> DA_data( A_data, A_data+size );
    thrust::device_vector<float> D_x( x, x+col );
    thrust::device_vector<float> D_b( b, b+row );

    float norm = 0.0f;

    //solve equation
    {
        hipsolverSpHandle_t handle;
        hipsolverStatus_t cusolver_status = hipsolverSpCreate(&handle);
        status_check( cusolver_status );
        //assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS && "failed to initialize cusolver");

        hipsparseMatDescr_t descrA;
        hipsparseStatus_t cusparse_status = cusparse_status = hipsparseCreateMatDescr(&descrA);
        assert(cusparse_status == HIPSPARSE_STATUS_SUCCESS && "failed to create Matrix descriptor");

        hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);

        float tol = 1.0e-7f;
        thrust::device_vector<int> D_p( row );
        int rankA = 0;

        std::cout << "solver: solver created." << std::endl;

        //convet COO matrix to CSR
        thrust::device_vector<int> DA_csr_row( row+1 );
        {
            hipsparseHandle_t handle=0;
            hipsparseStatus_t status= hipsparseCreate(&handle);
            assert( status == HIPSPARSE_STATUS_SUCCESS && "failed to initialize cusparse library" );

            hipsparseStatus_t error = hipsparseXcoo2csr( handle, DA_row.data().get(), size, row, DA_csr_row.data().get(), HIPSPARSE_INDEX_BASE_ONE );
            assert( error == HIPSPARSE_STATUS_SUCCESS && "failed to convert coo matrix to csr matrix" );

            status = hipsparseDestroy(handle);
            assert( status == HIPSPARSE_STATUS_SUCCESS && "failed to release cusparse library" );

            std::cout << "COO matrix -> CSR matrix finished." << std::endl;
        }

        // solving
        //              Ax = b
        for (;;)
        {
            std::cout << "Solving equation with tol " << tol << std::endl;

            cusolver_status = cusolverSpScsrlsqvqrHost(
                                                        //// Input
                                                        handle,                 // handle to the cuSolver library context
                                                        row,                    // number of rows
                                                        col,                    // number of columns
                                                        size,                   // number of non-zeros
                                                        descrA,                 // description of matrix
                                                        DA_data.data().get(),   // non-zero elements of the sparse matrix
                                                        DA_csr_row.data().get(),// row index of CSR matrix
                                                        DA_col.data().get(),    // column index of CSR matrix
                                                        D_b.data().get(),       // right handd data of b
                                                        tol,                    // tolerance to decide rank of A
                                                        //// Output
                                                        &rankA,                 // numerical rank of A
                                                        D_x.data().get(),       // solution of x
                                                        D_p.data().get(),       // permutation matrix
                                                        &norm                   // ||Ax-b||
                                                      );

            status_check( cusolver_status );

            tol *= 10.0f;

            if (cusolver_status == HIPSOLVER_STATUS_SUCCESS )
                break;

            if ( tol > 1.0f )
                assert( !"Failed to optimize the LLSF problem." );
        }
    }

    //copy D_x to x
    thrust::copy( D_x.begin(), D_x.end(), x );

    return norm;
}





