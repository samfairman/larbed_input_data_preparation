#include "hip/hip_runtime.h"
#include <f/device/assert/cuda_assert.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <cassert>

struct cuda_pattern;

cuda_pattern* make_cuda_pattern( unsigned long n, unsigned long ug_size );
void release_cuda_pattern( cuda_pattern* cp );
void cuda_pattern_register_entry( cuda_pattern* cp, unsigned long index, unsigned long dim, unsigned long* ar, double* diag, double* intensity );
void cuda_pattern_update_ug_thickness( cuda_pattern* cp, double* p );
double cuda_pattern_make_residual( cuda_pattern* cp );
double cuda_pattern_make_residual( cuda_pattern* cp );

//device
struct individual_pattern
{
    //all gpu
    unsigned long dim;
    double* ug_thickness; //ug_size * 2 + 1
    unsigned long* ar;
    double* diag;
    double* I_exp;
    double* I_sim;
    double2* A;
    double2* S;
    double2* S0;
    double2* S1;
    double2* S2;
    double2* S3;
    double2* S4;
    double2* S5;
    double2* S6;
    double2* S7;
    double2* S8;
    double2* S9;
    double2* S10;
};

//host
struct pattern_on_gpu
{
    int gpu_id;
    int pattern_id_start;
    int pattern_id_end;
    individual_pattern* ip; //cpu -- > gpu ...

    double* ug_thickness; //gpu
};

//host
struct cuda_pattern
{
    unsigned long total_ug_size; //--> count as complex type
    unsigned long total_cuda_pattern;
    int total_gpu;
    pattern_on_gpu* pog; //cpu
};

void cuda_pattern_update_ug_thickness( cuda_pattern* cp, double* p )
{
    //copy to gpu
    for ( int index = 0; index != (*cp).total_gpu; ++index )
    {
        //!!GPU CODE
        cuda_assert( hipSetDevice(index) );
        cuda_assert( hipMemcpy( ((*cp).pog)[index].ug_thickness, p, sizeof(unsigned long)*dim*dim, hipMemcpyHostToDevice ) );

    }
}

void release_cuda_pattern( cuda_pattern* cp )
{
    for ( int index = 0; index != (*cp).total_gpu; ++index )
    {
        pattern_on_gpu* the_pog = (*cp).pog + index; 
        
        //switch to gpu
        //!!GPU CODE
        cuda_assert( hipSetDevice((*the_pog).gpu_id) );
        cuda_assert( hipFree( (*the_pog).ug_thickness ) );
        (*the_pog).ug_thickness = 0;

        int const total_individual_pattern = (*the_pog).pattern_id_end - (*the_pog).pattern_id.start;

        for ( int jndex = 0; jndex != total_individual_pattern; ++jndex )
        {
            individual_pattern* the_individual_pattern = (*the_pog).ip + jndex;

            //cuda_assert( hipFree( (*the_individual_pattern).ug_thickness ) );
            cuda_assert( hipFree( (*the_individual_pattern).ar ) );
            cuda_assert( hipFree( (*the_individual_pattern).diag ) );
            cuda_assert( hipFree( (*the_individual_pattern).I_exp ) );
            cuda_assert( hipFree( (*the_individual_pattern).I_sim ) );
            cuda_assert( hipFree( (*the_individual_pattern).A ) );
            cuda_assert( hipFree( (*the_individual_pattern).S ) );
            cuda_assert( hipFree( (*the_individual_pattern).S0 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S1 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S2 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S3 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S4 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S5 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S6 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S7 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S8 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S9 ) );
            cuda_assert( hipFree( (*the_individual_pattern).S10 ) );

            //(*the_individual_pattern).ug_thickness = 0;
            (*the_individual_pattern).ar = 0;
            (*the_individual_pattern).diag = 0;
            (*the_individual_pattern).I_exp = 0;
            (*the_individual_pattern).I_sim = 0;
            (*the_individual_pattern).A = 0;
            (*the_individual_pattern).S = 0;
            (*the_individual_pattern).S0 = 0;
            (*the_individual_pattern).S1 = 0;
            (*the_individual_pattern).S2 = 0;
            (*the_individual_pattern).S3 = 0;
            (*the_individual_pattern).S4 = 0;
            (*the_individual_pattern).S5 = 0;
            (*the_individual_pattern).S6 = 0;
            (*the_individual_pattern).S7 = 0;
            (*the_individual_pattern).S8 = 0;
            (*the_individual_pattern).S9 = 0;
            (*the_individual_pattern).S10 = 0;
        }

        delete[] (*the_pog).ip;
    }

    delete[] (*cp).pog;
    (*cp).pog = 0;

    delete cp;
    cp = 0;
}

void cuda_pattern_register_entry( cuda_pattern* cp, unsigned long index, unsigned long dim, unsigned long* ar, double* diag, double* intensity )
{
    //find coresponding individual_pattern
    int gpu_index = 0;
    for ( gpu_index = 0; gpu_index != (*cp).total_gpu; ++gpu_index )
        if ( index < ((*cp).pog)[gpu_index].pattern_id_end ) 
            break;

    pattern_on_gpu* the_pog = (*cp).pog + gpu_index;
    int const pattern_index = index - (*the_pog).pattern_id_start;
    individual_pattern* the_individual_pattern = (*the_pog).ip + pattern_index;

    //!!GPU CODE

    //switch to the current gpu
    cuda_assert( hipSetDevice(gpu_index) );

    //allocate memory
    //cuda_assert( hipMalloc( &( (*the_individual_pattern).ug_thickness ), sizeof(double) * ( (*cp).total_ug_size * 2 + 1 ) ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).ar ), sizeof(unsigned long) * dim * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).diag ), sizeof(double) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).I_exp ), sizeof(double) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).I_sim ), sizeof(double) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).A ), sizeof(double) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S0 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S1 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S2 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S3 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S4 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S5 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S6 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S7 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S8 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S9 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*the_individual_pattern).S10 ), sizeof(double2) * dim ) );

    //copy ar
    cuda_assert( hipMemcpy( (*the_individual_pattern).ar, ar, sizeof(unsigned long)*dim*dim, hipMemcpyHostToDevice ) );
    //copy diag
    cuda_assert( hipMemcpy( (*the_individual_pattern).diag, diag, sizeof(double)*dim, hipMemcpyHostToDevice ) );
    //copy intensity
    cuda_assert( hipMemcpy( (*the_individual_pattern).I_exp, intensity, sizeof(double)*dim, hipMemcpyHostToDevice ) );
}

cuda_pattern* make_cuda_pattern( unsigned long n, unsigned long ug_size; )
{
    int total_gpus_avaliable = 0;
    cuda_assert( hipGetDeviceCount(&total_gpus_avaliable) );
    assert( total_gpus_avaliable > 0 );
    int const patterns_per_gpu = ( n + total_gpus_avaliable - 1 ) / total_gpus_avaliable;
    int const patterns_on_last_gpu = n - patterns_per_gpu * ( total_gpus_avaliable - 1 );

    //cuda_pattern
    cuda_pattern* cp = new cuda_pattern;
    (*cp).total_ug_size = ug_size;
    (*cp).total_gpu = total_gpus_avaliable;
    (*cp).total_cuda_pattern = n;
    (*cp).pog = new pattern_on_gpu[total_gpus_avaliable];
    
    //pattern on gpu
    for ( int index = 0; index != total_gpus_avaliable - 1; ++index ) //will skip if only one gpu
    {
        ((*cp).pog)[index].gpu_id = index;
        ((*cp).pog)[index].pattern_id_start = index * pattern_on_gpu;
        ((*cp).pog)[index].pattern_id_end = ( index + 1 ) * pattern_on_gpu;
        ((*cp).pog)[index].ip = new individual_pattern[patterns_per_gpu];


        //!!GPU CODE
        cuda_assert( hipSetDevice(index) );
        cuda_assert( hipMalloc( &( ((*cp).pog)[index].ug_thickness ), sizeof(double) * ( (*cp).total_ug_size * 2 + 1 ) ) );
    }
    
    if ( patterns_on_last_gpu > 0 ) //the last gpu
    {
        int const index = total_gpus_avaliable - 1;
        ((*cp).pog)[index].gpu_id = index;
        ((*cp).pog)[index].pattern_id_start = index * pattern_on_gpu;
        ((*cp).pog)[index].pattern_id_end = ( index + 1 ) * pattern_on_gpu;
        ((*cp).pog)[index].ip = new individual_pattern[patterns_per_gpu];

        //!!GPU CODE
        cuda_assert( hipSetDevice(index) );
        cuda_assert( hipMalloc( &( ((*cp).pog)[index].ug_thickness ), sizeof(double) * ( (*cp).total_ug_size * 2 + 1 ) ) );
    }
    
    //individual_pattern


    retun cp;
}

