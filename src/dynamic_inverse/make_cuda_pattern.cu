#include "hip/hip_runtime.h"
#include "cuda_pattern.h"

#include <f/device/assert/cuda_assert.hpp>

#include <cassert>

void make_individual_pattern( int gpu_id, individual_pattern* ip, unsigned long dim, unsigned long* ar, double* diag, double* intensity, double* ug_thickness )
{
    assert( gpu_id >= 0 );
    assert( (*ip).dim = 0 );
    assert( !(*ip).ug_thickness );
    assert( !(*ip).ar );
    assert( !(*ip).diag );
    assert( !(*ip).I_exp );
    assert( !(*ip).I_sim );
    assert( !(*ip).A );
    assert( !(*ip).S );
    assert( !(*ip).S0 );
    assert( !(*ip).S1 );
    assert( !(*ip).S2 );
    assert( !(*ip).S3 );
    assert( !(*ip).S4 );
    assert( !(*ip).S5 );
    assert( !(*ip).S6 );
    assert( !(*ip).S7 );
    assert( !(*ip).S9 );
    assert( !(*ip).S10 );
    assert( ip );
    assert( dim );
    assert( ar );
    assert( diag );
    assert( intensity );
    assert( ug_thickness );

    (*ip).ug_thickness = ug_thickness;
    (*ip).dim = dim;

    cuda_assert( hipSetDevice(gpu_index) );

    cuda_assert( hipMalloc( &( (*ip).ar ), sizeof(unsigned long) * dim * dim ) );
    cuda_assert( hipMalloc( &( (*ip).diag ), sizeof(double) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).I_exp ), sizeof(double) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).I_sim ), sizeof(double) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).A ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S0 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S1 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S2 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S3 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S4 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S5 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S6 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S7 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S8 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S9 ), sizeof(double2) * dim ) );
    cuda_assert( hipMalloc( &( (*ip).S10 ), sizeof(double2) * dim ) );

    //copy ar
    cuda_assert( hipMemcpy( (*ip).ar, ar, sizeof(unsigned long)*dim*dim, hipMemcpyHostToDevice ) );
    //copy diag
    cuda_assert( hipMemcpy( (*ip).diag, diag, sizeof(double)*dim, hipMemcpyHostToDevice ) );
    //copy intensity
    cuda_assert( hipMemcpy( (*ip).I_exp, intensity, sizeof(double)*dim, hipMemcpyHostToDevice ) );
}

void make_individual_pattern( individual_pattern* ip )
{
    (*ip).dim = 0;
    (*ip).ug_thickness = 0;
    (*ip).ar = 0;
    (*ip).diag = 0;
    (*ip).I_exp = 0;
    (*ip).I_sim = 0;
    (*ip).A = 0;
    (*ip).S = 0;
    (*ip).S0 = 0;
    (*ip).S1 = 0;
    (*ip).S2 = 0;
    (*ip).S3 = 0;
    (*ip).S4 = 0;
    (*ip).S5 = 0;
    (*ip).S6 = 0;
    (*ip).S7 = 0;
    (*ip).S8 = 0;
    (*ip).S9 = 0;
    (*ip).S10 = 0;
}

void make_pattern_on_gpu( pattern_on_gpu* pog, int const gpu_id_, int const pattern_id_start_, int const pattern_id_end_  )
{
    assert( pog );
    assert( pattern_id_end_ > pattern_id_start_ );

    unsigned long const total_ip = pattern_id_end_ - pattern_id_start_;

    (*pog).gpu_id = gpu_id_;
    (*pog).pattern_id_start = pattern_id_start_;
    (*pog).pattern_id_end = pattern_id_end_;

    (*pog).ip = new individual_pattern[total_ip];
    for ( unsigned long index = 0; index != total_ip; ++index )
        make_individual_pattern( (*pog).ip + index );

    (*pog).ip_cpu = 0;
    (*pog).ug_thickness = 0;
}

/**
  * n           --      total_tilt
  * ug_size     --      reflections  
  */
cuda_pattern* make_cuda_pattern( unsigned long n, unsigned long ug_size; )
{
    assert( n );
    assert( ug_size );

    //alloc patterns on gpu
    int total_gpus_avaliable = 0;
    cuda_assert( hipGetDeviceCount(&total_gpus_avaliable) );
    assert( total_gpus_avaliable > 0 );
    int const patterns_per_gpu = ( n + total_gpus_avaliable - 1 ) / total_gpus_avaliable;  // --> on first M-1 gpu
    int const patterns_on_last_gpu = n - patterns_per_gpu * ( total_gpus_avaliable - 1 );  // --> on last gpu

    //cuda_pattern
    cuda_pattern* cp = new cuda_pattern;
    (*cp).total_ug_size = ug_size;
    (*cp).total_cuda_pattern = n;
    (*cp).total_gpu = total_gpus_avaliable;
    (*cp).pog = new pattern_on_gpu[total_gpus_avaliable];

    for ( int index = 0; index != total_gpus_avaliable; ++index )
    {
        int const start_pattern = index * patterns_per_gpu;
        int const end_pattern = start_pattern+patterns_per_gpu > n ? n : start_pattern+patterns_per_gpu;

        make_pattern_on_gpu( (*cp).pog+index, index, start_pattern, end_pattern );
    }

    return cp;

/*
    
    //pattern on gpu
    for ( int index = 0; index != total_gpus_avaliable - 1; ++index ) //will skip if only one gpu
    {
        ((*cp).pog)[index].gpu_id = index;
        ((*cp).pog)[index].pattern_id_start = index * pattern_on_gpu;
        ((*cp).pog)[index].pattern_id_end = ( index + 1 ) * pattern_on_gpu;
        ((*cp).pog)[index].ip = new individual_pattern[patterns_per_gpu];


        //!!GPU CODE
        cuda_assert( hipSetDevice(index) );
        cuda_assert( hipMalloc( &( ((*cp).pog)[index].ug_thickness ), sizeof(double) * ( (*cp).total_ug_size * 2 + 1 ) ) );

        for ( unsigned long jndex = 0; jndex != patterns_per_gpu; ++jndex )
            (((*cp).pog)[index].ip)[jndex] = ((*cp).pog)[index].ug_thickness;

        //ip_gpu set to null
        ((*cp).pog)[index].ip_gpu = 0;
    }
    
    if ( patterns_on_last_gpu > 0 ) //the last gpu
    {
        int const index = total_gpus_avaliable - 1;
        ((*cp).pog)[index].gpu_id = index;
        ((*cp).pog)[index].pattern_id_start = index * pattern_on_gpu;
        ((*cp).pog)[index].pattern_id_end = ( index + 1 ) * pattern_on_gpu;
        ((*cp).pog)[index].ip = new individual_pattern[patterns_on_last_gpu];

        //!!GPU CODE
        cuda_assert( hipSetDevice(index) );
        cuda_assert( hipMalloc( &( ((*cp).pog)[index].ug_thickness ), sizeof(double) * ( (*cp).total_ug_size * 2 + 1 ) ) );

        for ( unsigned long jndex = 0; jndex != patterns_on_last_gpu; ++jndex )
            (((*cp).pog)[index].ip)[jndex] = ((*cp).pog)[index].ug_thickness;

        //ip_gpu set to null
        ((*cp).pog)[index].ip_gpu = 0;
    }

    retun cp;
*/
}

