#include "hip/hip_runtime.h"
#include <f/device/device_assert/cuda_assert.hpp>
#include <f/device/device_assert/cublas_assert.hpp>
#include <f/device/device_assert/kernel_assert.hpp>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <hip/hip_complex.h>
#include <math_functions.h>

__global__ void Dznrm2( unsigned long m, double2 *dA, double *dxnorm )// Dznrm2<<<1,128>>>(...)
{
    unsigned long i = threadIdx.x;

    __shared__ double x[128];

    double lsum = 0.0;

    for( unsigned long j = i; j < m; j += 128 ) 
    {
        double const re = dA[j].x;
        double const im = dA[j].y;
        lsum += re*re + im*im;
    }

    x[i] = lsum;

    __syncthreads();

    if ( i <   64 ) { x[i] += x[i+  64]; }  __syncthreads();
    if ( i <   32 ) { x[i] += x[i+  32]; }  __syncthreads();
    if ( i <   16 ) { x[i] += x[i+  16]; }  __syncthreads();
    if ( i <    8 ) { x[i] += x[i+   8]; }  __syncthreads();
    if ( i <    4 ) { x[i] += x[i+   4]; }  __syncthreads();
    if ( i <    2 ) { x[i] += x[i+   2]; }  __syncthreads();
    if ( i <    1 ) { x[i] += x[i+   1]; }  __syncthreads();

    if ( i == 0 ) *dxnorm = sqrt(x[0]);
}

__device__ void device_Dznrm2( unsigned long m, double2 *dA, double *dxnorm )
{
    double ans = 0.0;
    for ( unsigned long index = 0; index != m; ++index )
    {
        double const real = dA[index].x;
        double const imag = dA[index].y;
        ans += real*real + imag*imag;
    }

    dxnorm[0] = ans;
}

__global__ void Dasum( unsigned long m, double2 *dA, double *dxnorm )
{
    unsigned long i = threadIdx.x;

    __shared__ double x[128];

    double lsum = 0.0;

    for( unsigned long j = i; j < m; j += 128 ) 
    {
        double const re = dA[j].x;
        double const im = dA[j].y;
        lsum += sqrt(re*re + im*im);
    }

    x[i] = lsum;

    __syncthreads();

    if ( i <   64 ) { x[i] += x[i+  64]; }  __syncthreads();
    if ( i <   32 ) { x[i] += x[i+  32]; }  __syncthreads();
    if ( i <   16 ) { x[i] += x[i+  16]; }  __syncthreads();
    if ( i <    8 ) { x[i] += x[i+   8]; }  __syncthreads();
    if ( i <    4 ) { x[i] += x[i+   4]; }  __syncthreads();
    if ( i <    2 ) { x[i] += x[i+   2]; }  __syncthreads();
    if ( i <    1 ) { x[i] += x[i+   1]; }  __syncthreads();

    if ( i == 0 ) *dxnorm = x[0];
}

//should call with Zscale<<<1, 128>>>(...);
__global__ void Zscal( unsigned long m, double real, double2* dA )
{
    const int i = threadIdx.x;

    for( unsigned long j = i; j < m; j += 128 ) 
    {
        dA[j].x *= real;
        dA[j].y *= real;
    }
}

__device__ void device_Zscal( unsigned long m, double real, double2* dA )
{
    //for ( unsigned long index = 0; index != m; ++index ) <<-- WHY this one doesnot work????????????????????????????????????????????????????????????????????????????????????????????????????????????????????????????????????????
    for ( unsigned long index = 0; index < m; ++index )
    {
        dA[index].x *= real;
        dA[index].y *= real;
    }
}

    //TODO: optimization
    __global__ //<<<((dim+15)/16,(dim+15)/16), (16,16)>>>
    void Zgemm( double2* P, double2* M, double2* N, unsigned long dim, double alpha )
    {
        typedef double              value_type;
        typedef double2             complex_type;
        typedef unsigned long       size_type;

        __shared__ value_type _M[16][17];
        __shared__ value_type _m[16][17];
        __shared__ value_type _N[16][17];
        __shared__ value_type _n[16][17];

        const size_type bx = blockIdx.x;
        const size_type by = blockIdx.y;
        const size_type tx = threadIdx.x;
        const size_type ty = threadIdx.y;
        const size_type row = by * 16 + ty;
        const size_type col = bx * 16 + tx;
        const size_type iter_n = (dim+15)/16;

        value_type R = 0.0;
        value_type I = 0.0;

        for ( size_type i = 0; i != iter_n; ++i )
        {
            if ( i * 16 + tx < dim && row < dim )
            {
                _M[ty][tx] = (*( M + row * dim + i * 16 + tx )).x;
                _m[ty][tx] = (*( M + row * dim + i * 16 + tx )).y;
            }
            else
            {
                _M[ty][tx] = 0.0;
                _m[ty][tx] = 0.0;
            }

            if ( i * 16 + ty < dim && col < dim )
            {
                _N[ty][tx] = (*( N + ( i * 16 + ty ) * dim + col )).x;
                _n[ty][tx] = (*( N + ( i * 16 + ty ) * dim + col )).y;
            }
            else
            {
                _N[ty][tx] = 0.0;
                _n[ty][tx] = 0.0;
            }

            __syncthreads();

            #pragma unroll
            for ( size_type j = 0; j != 16; ++j )
            {
                R += _M[ty][j] * _N[j][tx] - _m[ty][j] * _n[j][tx];
                I += _M[ty][j] * _n[j][tx] + _m[ty][j] * _N[j][tx];
            }
            __syncthreads();
        }

        if ( row < dim && col < dim )
        {
            (*( P + row * dim + col )).x = alpha * R;
            (*( P + row * dim + col )).y = alpha * I;
        }
    }

__global__ void //<<<1,128>>>
Zcopy( unsigned long dims, double2* src, double2* dst )
{
    unsigned long const i = threadIdx.x;

    for( unsigned long j = i; j < dims; j += 128 ) 
    {
        (*(dst+j)).x = (*(src+j)).x;
        (*(dst+j)).y = (*(src+j)).y;
    }
}

__device__ void
device_Zcopy( unsigned long dims, double2* src, double2* dst )
{
    for ( unsigned long index = 0; index < dims; ++index )
    {
        dst[index].x = src[index].x;
        dst[index].y = src[index].y;
    }
}

__global__ void//<<<1, 128>>>
Zaxpy( unsigned long dims, double real, double imag, double2* dst, double2* src ) // dst += (real,imag) * src
{
    unsigned long const i = threadIdx.x;
    double R = 0.0;
    double I = 0.0;

    for( unsigned long j = i; j < dims; j += 128 ) 
    {
        R = (*(src+j)).x;
        I = (*(src+j)).y;

        (*(dst+j)).x += real * R - imag * I;
        (*(dst+j)).y += real * I + imag * R;
    }
}

__device__ void//<<<1, 128>>>
device_Zaxpy( unsigned long dims, double real, double imag, double2* dst, double2* src ) // dst += (real,imag) * src
{
    for ( unsigned long index = 0; index < dims; ++index )
    {
        double const R = src[index].x;
        double const I = src[index].y;
        dst[index].x = real * R - imag * I;
        dst[index].y = real * I + imag * R;
    }
}
#if 0
__global__ void
compose_a( double* ug, unsigned long* ar, double* diag, double thickness, double2* a, unsigned long dim )
{
    int const row_index = threadIdx.x;

    for ( unsigned long col_index = 0; col_index != dim; ++col_index )
    {
        unsigned long a_offset = row_index * dim + col_index;
        unsigned long const ug_index = *(ar+a_offset);
        *(a+a_offset) = make_hipDoubleComplex( -thickness * (*(ug+ug_index+ug_index+1)), thickness *( *(ug+ug_index+ug_index)) );
    }

    *(a+row_index*dim+row_index) = make_hipDoubleComplex( 0.0, thickness *( *(diag+row_index) ) );
}
#endif
#if 0
__device__ void
device_compose_a( double* ug, unsigned long* ar, double* diag, double thickness, double2* a, unsigned long dim, double alpha, double beta, double gamma, double* beams )
{
    thickness *= 100.0;
    for ( unsigned long row_index = 0; row_index != dim; ++row_index )
    {
        for ( unsigned long col_index = 0; col_index != dim; ++col_index )
        {
            unsigned long a_offset = row_index * dim + col_index;
            unsigned long const ug_index = *(ar+a_offset);
            *(a+a_offset) = make_hipDoubleComplex( -thickness * (*(ug+ug_index+ug_index+1)), thickness *( *(ug+ug_index+ug_index)) );
        }
        unsigned long const beams_index = ar[row_index*dim];
        double const kx = beams[beams_index*10+1];
        double const ky = beams[beams_index*10+2];

        *(a+row_index*dim+row_index) = make_hipDoubleComplex( 0.0, thickness *( *(diag+row_index) + kx*alpha + ky*beta + gamma ) );
    }
}
#endif
__device__ void
device_compose_a( double* ug, unsigned long* ar, double* diag, double thickness, double2* a, unsigned long dim, double alpha, double beta, double* beams )
{
    thickness *= 100.0;
    for ( unsigned long row_index = 0; row_index != dim; ++row_index )
    {
        for ( unsigned long col_index = 0; col_index != dim; ++col_index )
        {
            unsigned long a_offset = row_index * dim + col_index;
            unsigned long const ug_index = *(ar+a_offset);
            *(a+a_offset) = make_hipDoubleComplex( -thickness * (*(ug+ug_index+ug_index+1)), thickness *( *(ug+ug_index+ug_index)) );
        }
        unsigned long const beams_index = ar[row_index*dim];
        double const kx = beams[beams_index*10+1];
        double const ky = beams[beams_index*10+2];

        *(a+row_index*dim+row_index) = make_hipDoubleComplex( 0.0, thickness * ( *(diag+row_index) + kx*alpha + ky*beta ) );
    }
}

//TODO: optimization
#if 0
Comment:
            When working with original global kernel 'extract_intensity_diff_with_offset_zigmoid', the generated residuals( all kinds ) are
            a little bit smaller(1.0e-6 order) than the new device routine 'device_extract_intensity_diff_with_offset_zigmoid'
#endif
__global__ void
extract_intensity_diff_with_offset_zigmoid( double2* s, double* I_exp, double* I_diff, double* I_zigmoid, unsigned long dim, unsigned long column_index, double ac_offset, double dc_offset, double c )
{
    int const I_offset = threadIdx.x;
    int const S_offset = column_index + threadIdx.x * dim;
    double const norm = hipCabs(*(s+S_offset));

    double const val = *(I_exp+I_offset);
    double const df = val - norm * norm * ac_offset - dc_offset;
    *(I_diff+I_offset) = df;
    *(I_zigmoid+I_offset) = df / ( 1.0 + exp( 12.56637061435917295384*c*val ) );
}

__device__ void
device_extract_intensity_diff_with_offset_zigmoid( double2* s, double* I_exp, double* I_diff, double* I_zigmoid, unsigned long dim, unsigned long column_index, double ac_offset, double dc_offset, double c )
{
    for ( unsigned long index = 0; index < dim; ++index )
    {
        unsigned long const I_offset = index;
        unsigned long const S_offset = column_index + index * dim;
        double const real = s[S_offset].x;
        double const imag = s[S_offset].y;
        double const norm = real*real + imag*imag;
        double const val  = I_exp[I_offset];
        double const df = val - norm * ac_offset - dc_offset;
        I_diff[I_offset] = df;
        I_zigmoid[I_offset] = df / ( 1.0 + exp( 12.56637061435917295384*c*val ) );
    }
}

//TODO: optimization
__global__ void
sum_diag( double2* a, unsigned long dim, double real, double imag )
{
    int const index = threadIdx.x;
    int const offset = index * dim + index;
    *(a+offset) = make_hipDoubleComplex( hipCreal(*(a+offset))+real, hipCimag(*(a+offset))+imag );
}

__device__ void
device_sum_diag( double2* a, unsigned long dim, double real, double imag )
{
    for ( unsigned long index = 0; index < dim; ++index )
    {
        unsigned long const offset = index * dim + index;
        a[offset].x += real;
        a[offset].y += imag;
    }
}

/*
 * Input/Output:
 *
 ** ug[M]
 *  ar[n][n]
 *  diag[n]         ==>>    I_diff[n]
 ** thickness
 *  dim -- n
 *  I_exp[n]
 ** column_index
 *
 *  cache:
 *  a_[n][n]    -- p2p3
 *  a^2_[n][n]  -- s
 *  a^3_[n][n]  -- s_
 *  P1[n][n]
 *  P2[n][n]
 *  P3[n][n]
 *
 * 1) compose A
 * 2) scale to A_
 * 3) compute A_^2 A_^3
 * 4) compute (P1) (P2) (P3)
 * 5) square back
 * 6) extract one column
 */
__global__ void
make_individual_pattern_intensity_diff( double* cuda_ug, unsigned long* cuda_ar, double* cuda_diag, double thickness, unsigned long* cuda_dim, double* cuda_I_exp, double* cuda_I_diff, unsigned long column_index, double2* cuda_cache, unsigned long max_dim, unsigned long tilt_size, double c, double * cuda_I_zigmoid, double* beams, double* kt_factor )
{
    unsigned long const tilt_index = blockDim.x * blockIdx.x + threadIdx.x;

    if ( tilt_index >= tilt_size ) return;

    unsigned long const dim = *(cuda_dim + tilt_index);
    double* ug = cuda_ug;
    unsigned long* ar = cuda_ar + tilt_index * max_dim * max_dim;
    double* diag = cuda_diag + tilt_index * max_dim;
    double* I_exp = cuda_I_exp + tilt_index * max_dim;
    double* I_diff = cuda_I_diff + tilt_index * max_dim;
    double* I_zigmoid = cuda_I_zigmoid + tilt_index * max_dim;
    double2* cache = cuda_cache + 6 * tilt_index * max_dim * max_dim;

    unsigned long dimdim = dim*dim;

    //cache should be of size 6*N^2
    double2* a_ = cache;
    double2* aa_ = a_ + dimdim;
    double2* aaa_ = aa_ + dimdim;
    double2* p1 = aaa_ + dimdim;
    double2* p2 = p1 + dimdim;
    double2* p3 = p2 + dimdim;

    //reuse memory in latter steps, when a_, aa_ and aaa_ are idle
    //double2* p2p3 = a_;
    double2* p2p3 = aaa_;
    double2* s = aa_;
    double2* s_ = aaa_;

    //1)
    //kernel_assert( (compose_a<<<1, dim>>>( ug, ar, diag, thickness, a_, dim )) );
    //cuda_assert( hipDeviceSynchronize() );
    //device_compose_a( ug, ar, diag, thickness, a_, dim );

    //double const alpha = kt_factor[tilt_index*3];
    //double const beta = kt_factor[tilt_index*3+1];
    //double const gamma = kt_factor[tilt_index*3+2];
    //device_compose_a( ug, ar, diag, thickness, a_, dim, alpha, beta, gamma, beams );

    double const alpha = kt_factor[tilt_index*2];
    double const beta = kt_factor[tilt_index*2+1];
    device_compose_a( ug, ar, diag, thickness, a_, dim, alpha, beta, beams );

    //2)
    //TODO
    double* the_norm = (double*)aa_;
    //kernel_assert( (Dznrm2<<<1,128>>>( dimdim, a_, the_norm )) );
    ////kernel_assert( (Dasum<<<1,128>>>( dimdim, a_, the_norm )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Dznrm2( dimdim, a_, the_norm );

    //double const ratio = (*the_norm) * 53.71920351148152;
    double const ratio = (*the_norm) / 5.371920351148152;
    unsigned long const scaler = ratio < 1.0 ? 0 : ceil(log2(ratio));
    unsigned long const scaling_factor =  1 << scaler;
    double const scale = scaling_factor;
    //kernel_assert( (Zscal<<<1, 128>>>( dimdim, 1.0/scale, a_ )) );    //a_ /= scale
    //cuda_assert( hipDeviceSynchronize() );
    device_Zscal( dimdim, 1.0/scale, a_ );

    //3)
    dim3 const mm_grids( (dim+15)/16, (dim+15)/16 );
    dim3 const mm_threads( 16, 16 );
    kernel_assert( (Zgemm<<<mm_grids, mm_threads>>>( aa_, a_, a_, dim, 1.0 )) );
    cuda_assert( hipDeviceSynchronize() );
    kernel_assert( (Zgemm<<<mm_grids, mm_threads>>>( aaa_, aa_, a_, dim, 1.0 )) );
    cuda_assert( hipDeviceSynchronize() );

    //4)
    /*
     * Maple:
     *  Digits := 25
     *  evalf(solve(_Z^9+9*_Z^8+72*_Z^7+504*_Z^6+3024*_Z^5+15120*_Z^4+60480*_Z^3+181440*_Z^2+362880*_Z+362880 = 0))
     * Returns:
     *  2.697333461536989227389605+5.184162062649414177834087*I,     //c1
     *  -.3810698456631129990312942+4.384644533145397950369203*I,    //c2
     *  -2.110839800302654737498705+3.089910928725500922777702*I,    //c3
     *  -3.038648072936697089212469+1.586801195758838328803868*I,    //c4
     *  -3.333551485269048803294274,                                 //c5
     *  -3.038648072936697089212469-1.586801195758838328803868*I,    //c6
     *  -2.110839800302654737498705-3.089910928725500922777702*I,    //c7
     *  -.3810698456631129990312942-4.384644533145397950369203*I,    //c8
     *  2.697333461536989227389605-5.184162062649414177834087*I      //c9
     *
     *  expand((x-c1)*(x-c2)*(x-c3))  >> p1                                                                                                   (                     p1_c                             )
     *      x^3-.205423815571221490859606*x^2-(12.65871752452031305098099*I)*x^2-58.21460179641193947200471*x-(3.189848964212376356715960*I)*x-19.71085376106750328141397+94.20645646169128946503649*I
     *
     *  expand((x-c4)*(x-c5)*(x-c6))  >> p2   (         p2_c            )
     *      x^3+9.410847631142442981719212*x^2+39.17363072664900708597702-6.123261017392618755198919*10^(-24)*I+32.01029973951970099352671*x+(4.*10^(-24)*I)*x
     *
     *  expand((x-c7)*(x-c8)*(x-c9))  >> p3                                                                                                  (                         p3_c                         )
     *      x^3-.205423815571221490859601*x^2+(12.65871752452031305098099*I)*x^2-58.21460179641193947200470*x+(3.18984896421237635671600*I)*x-19.71085376106750328141404-94.20645646169128946503646*I
     *
     *  expand((x-c1)*(x-c2)*(x-c3)*(x-c4)*(x-c5)*(x-c6)*(x-c7)*(x-c8)*(x-c9))
     *      3.628800000000000000000003*10^5-1.365022562699469279472268*10^(-19)*I+3.628800000000000000000003*10^5*x+x^9+9.00000000000000000000000*x^8+72.00000000000000000000006*x^7+503.9999999999999999999995*x^6+3024.000000000000000000002*x^5+15120.00000000000000000000*x^4+60479.99999999999999999995*x^3+1.814400000000000000000001*10^5*x^2-(5.*10^(-22)*I)*x^6-(1.*10^(-20)*I)*x^4-(1.0*10^(-19)*I)*x^3+(2.*10^(-24)*I)*x^8-(3.0*10^(-19)*I)*x^2-(7.*10^(-21)*I)*x^5-(4.*10^(-19)*I)*x+(2.*10^(-23)*I)*x^7
     */
    //4 - p1)
    //kernel_assert( (Zcopy<<<1,128>>>( dimdim, aaa_, p1 )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zcopy( dimdim, aaa_, p1 );
    //kernel_assert( (Zaxpy<<<1,128>>>( dimdim, -0.205423815571221490859606, -12.65871752452031305098099, p1, aa_ )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zaxpy( dimdim, -0.205423815571221490859606, -12.65871752452031305098099, p1, aa_ );
    //kernel_assert( (Zaxpy<<<1,128>>>( dimdim, -58.21460179641193947200471, -3.189848964212376356715960, p1, a_ )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zaxpy( dimdim, -58.21460179641193947200471, -3.189848964212376356715960, p1, a_ );
    //kernel_assert( (sum_diag<<<1,dim>>>( p1, dim, -19.71085376106750328141397, 94.20645646169128946503649 )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_sum_diag( p1, dim, -19.71085376106750328141397, 94.20645646169128946503649 );

    //4 - p2)
    //kernel_assert( (Zcopy<<<1,128>>>( dimdim, aaa_, p2 )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zcopy( dimdim, aaa_, p2 );
    //kernel_assert( (Zaxpy<<<1,128>>>( dimdim, 9.410847631142442981719212, 0.0, p2, aa_ )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zaxpy( dimdim, 9.410847631142442981719212, 0.0, p2, aa_ );
    //kernel_assert( (Zaxpy<<<1,128>>>( dimdim, 32.01029973951970099352671, 0.0, p2, a_ )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zaxpy( dimdim, 32.01029973951970099352671, 0.0, p2, a_ );
    //kernel_assert( (sum_diag<<<1,dim>>>( p2, dim, 39.17363072664900708597702, 0.0  )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_sum_diag( p2, dim, 39.17363072664900708597702, 0.0 );

    //4 - p3)
    //kernel_assert( (Zcopy<<<1,128>>>( dimdim, aaa_, p3 )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zcopy( dimdim, aaa_, p3 );
    //kernel_assert( (Zaxpy<<<1,128>>>( dimdim, -0.205423815571221490859601, 12.65871752452031305098099, p3, aa_ )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zaxpy( dimdim, -0.205423815571221490859601, 12.65871752452031305098099, p3, aa_ );
    //kernel_assert( (Zaxpy<<<1,128>>>( dimdim, -58.21460179641193947200470, 3.18984896421237635671600, p3, a_ )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_Zaxpy( dimdim, -58.21460179641193947200470, 3.18984896421237635671600, p3, a_ );
    //kernel_assert( (sum_diag<<<1,dim>>>( p3, dim, -19.71085376106750328141404, -94.20645646169128946503646 )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_sum_diag( p3, dim, -19.71085376106750328141404, -94.20645646169128946503646 );

    //4 - s)
    kernel_assert( (Zgemm<<<mm_grids, mm_threads>>>( p2p3, p2, p3, dim, 0.0016600397351866578333 )) );
    cuda_assert( hipDeviceSynchronize() );
    kernel_assert( (Zgemm<<<mm_grids, mm_threads>>>( s, p1, p2p3, dim, 0.0016600397351866578333 )) );
    cuda_assert( hipDeviceSynchronize() );

    //5)
    if ( scaler != 0 )
    {
        for ( unsigned long index = 0; index != scaler; ++index )
        {
            kernel_assert( (Zgemm<<<mm_grids, mm_threads>>>( s_, s, s, dim, 1.0 )) );
            cuda_assert( hipDeviceSynchronize() );
            double2* tmp = s_;
            s_ = s;
            s = tmp;
        }
    }

    //6)
    double const ac_offset = cuda_ug[0];
    double const dc_offset = cuda_ug[1];
    //kernel_assert( (extract_intensity_diff_with_offset_zigmoid<<<1,dim>>>( s, I_exp, I_diff, I_zigmoid, dim, column_index, ac_offset, dc_offset, c )) );
    //cuda_assert( hipDeviceSynchronize() );
    device_extract_intensity_diff_with_offset_zigmoid( s, I_exp, I_diff, I_zigmoid, dim, column_index, ac_offset, dc_offset, c );
}

void make_pattern_intensity_diff( double* cuda_ug, unsigned long* cuda_ar, double* cuda_diag, double thickness, unsigned long* cuda_dim, double* cuda_I_exp, double* cuda_I_diff, unsigned long column_index, double2* cuda_cache, unsigned long tilt_size, unsigned long max_dim, double c, double* cuda_I_zigmoid, double* beams, double* kt_factor )
{
    //unsigned long const threads = 64;
    //unsigned long const threads = 128;
    unsigned long const threads = 256;
    unsigned long const grids = (tilt_size + threads - 1)/threads;

    kernel_assert( ( make_individual_pattern_intensity_diff<<<grids, threads>>>( cuda_ug, cuda_ar, cuda_diag, thickness, cuda_dim, cuda_I_exp, cuda_I_diff, column_index, cuda_cache, max_dim, tilt_size, c, cuda_I_zigmoid, beams, kt_factor ) ) );
    //cuda_assert( hipDeviceSynchronize() );
}

