#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math_functions.h>

typedef unsigned long   size_type;
typedef size_type*      size_pointer_type;
typedef double          value_type;
typedef value_type*     value_pointer_type;
typedef hipDoubleComplex complex_type;
typedef complex_type*   complex_pointer_type;

// aa = a * a
void __global__ zgemm_aa( complex_pointer_type __restrict__ a, complex_pointer_type __restrict__ aa, size_type n )
{
}

void __global__ zgemm_a_b_c( complex_pointer_type __restrict__ a, complex_pointer_type __restrict__ b, complex_pointer_type __restrict__ c, size_type n )
{
}

void __global__
direct_sd_update_(  size_type                                  align_dim_,
                    size_type                                  ug_size_,
                    size_pointer_type             __restrict__ dim_cache_, 
                    complex_pointer_type          __restrict__ ug_cache_,  
                    complex_pointer_type          __restrict__ S_cache_,
                    complex_pointer_type          __restrict__ diag_cache_,
                    value_pointer_type            __restrict__ I_new_cache_,
                    value_pointer_type            __restrict__ I_ori_cache_,
                    size_pointer_type             __restrict__ ar_cache_,
                    complex_pointer_type          __restrict__ A_cache_,
                    complex_pointer_type          __restrict__ a_cache_,
                    complex_pointer_type          __restrict__ aa_cache_,
                    complex_pointer_type          __restrict__ aaa_cache_,
                    complex_pointer_type          __restrict__ PP_cache_,
                    complex_pointer_type          __restrict__ PQ_cache_,
                    complex_pointer_type          __restrict__ PR_cache_,
                    complex_pointer_type          __restrict__ PS_cache_,
                    value_pointer_type            __restrict__ residual_
                )
{
    size_type const block_index   = blockIdx.x;
    size_type const thread_index  = threadIdx.x;
    size_type const total_task    = align_dim_ >> 5; //threads per block is set to 32 
    size_type const task_begin    = thread_index * total_task;
    size_type const task_end      = task_begin + total_task;
    size_type const real_dim      = dim_cache_[block_index];

    size_type const matrix_offset = block_index * align_dim_ * align_dim_;
    size_type const array_offset  = block_index * align_dim_;

    complex_pointer_type A        = A_cache_ + matrix_offset;
    complex_pointer_type S        = S_cache_ + matrix_offset;
    complex_pointer_type diag     = diag_cache_ + array_offset;
    value_pointer_type I_new      = I_new_cache_ + array_offset;
    value_pointer_type I_ori      = I_ori_cache_ + array_offset;
    size_pointer_type ar          = ar_cache_ + matrix_offset;
    complex_pointer_type a        = a_cache_ + matrix_offset;
    complex_pointer_type aa       = aa_cache_ + matrix_offset;
    complex_pointer_type aaa      = aaa_cache_ + matrix_offset;
    complex_pointer_type PP       = PP_cache_ + matrix_offset;
    complex_pointer_type PQ       = PQ_cache_ + matrix_offset;
    complex_pointer_type PR       = PR_cache_ + matrix_offset;
    complex_pointer_type PS       = PS_cache_ + matrix_offset;

    //update A in col major -- TODO in row major -- also calc norm_1
    size_type square_times = 0;
    value_type scaler = 1.0;

    {
        //load ug to shared memory
        __shared__ complex_type ug[ug_size_ + 1];
        {
            size_type const task_per_thread = (ug_size_ + 31) >> 5; //32 threads
            size_type const start = thread_index * task_per_thread;
            size_type const end   = start + task_per_thread;

            for ( size_type index = start; index != end; ++index ) //not need to check memory boundary
                ug[index] = ug_cache_[index];

            __syncthreads();
        }

        //construct A, reduce norm2 to shared memory
        __shared__ value_type norm_1[ align_dim_ ];
        for ( size_type col = task_begin; col != task_end; ++col )
        {
            norm_1[col] = 0.0;
            for ( size_type row = 0; row != align_dim_; ++row )
            {
                size_type const align_offset = row * align_dim_ + col;
                size_type const real_offset  = row * real_dim + col;
                complex_type const the_element = ug[ar[real_offset]]; //slow
                A[align_offset] = the_element;

                norm_1[col] += hypot( cuReal( the_element ) , cuImag( the_element ) );
            }

            size_type const align_offset = row * align_dim_ + col;
            complex_type const the_element = diag[col]
            A[align_offset] = the_element; 

            norm_1[col] += hypot( cuReal( the_element ) , cuImag( the_element ) );

            //reduce to the first element in norm_1
            norm_1[task_begin] = fmax( norm_1[task_begin], norm_1[col] );
        }
        __syncthreads();

        if ( 0 == thread_index )
            for ( size_type index = task_per_thread; index != align_dim_; index += task_per_thread )
                norm_1[0] = fmax( norm_1[0], norm_1[index] );

        __syncthreads();

        value_type the_norm_1 = sqrt( norm_1[0] );
        value_type ratio = 5.371920351148152 / the_norm_1;
        square_times =  ratio < 1.0 ? 0 : (size_type)( ceil( log2( ratio ) ) );
        scaler = ratio < 1.0 ? (value_type)(1 << s) : 1.0;
    }

    //scale A to a
    {
        for ( size_type col = task_begin; col != task_end; ++col )
            for ( size_type row = 0; row != align_dim_; ++row )
            {
                size_type const offset = row * align_dim_ + col;
                a[offset] = A[offset] / scaler;
            }
        __syncthreads();
    }

    //calc aa
    {
        size_type const grids = ( align_dim_ + 31 ) >> 5;
        dim3 const blocks{ 16, 16 };
        if ( 0 == thread_index )
            zgemm_aa<<< grids, blocks >>>( a, aa, align_dim_ );
        __syncthreads();
    }

    //calc aaa
    {
        size_type const grids = ( align_dim_ + 31 ) >> 5;
        dim3 const blocks{ 16, 16 };
        if ( 0 == thread_index )
            zgemm_a_b_c<<< grids, blocks >>>( a, aa, aaa, align_dim_ );
        __syncthreads();
    }

    //calc PP



    //calc PQ

    //calc PR

    //calc PS

    //calc PP PQ PR PS


    //scale back

    //update I new

    //make diff

}

void direct_sd_update(  size_type tilt_size_,
                        size_type align_dim_, 
                        size_type ug_size_,
                        size_pointer_type dim_cache_,
                        complex_pointer_type ug_cache_,
                        complex_pointer_type S_cache_,
                        complex_pointer_type diag_cache_,
                        value_pointer_type I_new_cache_,
                        value_pointer_type I_ori_cache_,
                        size_pointer_type ar_cache_,
                        complex_pointer_type A_cache_,
                        complex_pointer_type a_cache_,
                        complex_pointer_type aa_cache_,
                        complex_pointer_type aaa_cache_,
                        complex_pointer_type PP_cache_,
                        complex_pointer_type PQ_cache_,
                        complex_pointer_type PR_cache_,
                        complex_pointer_type PS_cache_,
                        value_pointer_type residual_
                    )
{
        unsigned long const block_size = tilt_size_;
        unsigned long const thread_size = 16;

        direce_sd_update_<<<block_size, thread_size>>>( align_dim_, ug_size_, dim_cache_, ug_cache_, S_cache_, diag_cache_, I_new_cache_, I_ori_cache_, ar_cache_, A_cache_, a_cache_, aa_cache_, aaa_cache_, PP_cache_, PQ_cache_, PR_cache_, PS_cache_, residual_);
}

