#include "./cuda_include.h"

void set_device( int id )
{
    int current_id;
    cuda_assert( hipGetDevice( &current_id ) );
    if ( current_id != id )
        cuda_assert( hipSetDevice( id ) );
}

void cuda_allocate( void** p, unsigned long n )
{
    cuda_assert( hipMalloc( p, n ) );
    cuda_assert( hipMemset( *p, 0, n ) );
}

void cuda_deallocate( void* p )
{
    cuda_assert( hipFree( p ) );
}

void cuda_memcopy_host_to_device( const void* src, unsigned long n, void* dst )
{
    cuda_assert( hipMemcpy( dst, src, n, hipMemcpyHostToDevice  ) );
}

void cuda_memcopy_device_to_host( const void* src, unsigned long n, void* dst )
{
    cuda_assert( hipMemcpy( dst, src, n, hipMemcpyDeviceToHost  ) );
}

double dot( double* x_begin, double* x_end, double* y_begin )
{
    hipblasHandle_t handle;
    cublas_assert( hipblasCreate(&handle) );
    unsigned long n = x_end - x_begin;
    double ans;
    cublas_assert( hipblasDdot( handle, n, x_begin, 1, y_begin, 1, &ans ) );
    cublas_assert( hipblasDestroy(handle) );
    return ans;
}

