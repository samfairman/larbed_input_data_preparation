#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <f/device/assert/kernel_assert.hpp>

//
// P = alpha_0 P0 + alpha_1 P1 + alpha_2 P2 + alpha_3 P3
//
void __global__
zmatrix_poly4_eval_kernel(unsigned long n, double2* P, double alpha_0, double2* P0, double alpha_1, double2* P1, double alpha_2, double2* P2, double alpha_3, double2* P3 )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;
    if ( tx < n )
    {
        (*(P+tx)).x = alpha_0 * ((*(P0+tx)).x) + alpha_1 * ((*(P1+tx)).x) + alpha_2 * ((*(P2+tx)).x) + alpha_3 * ((*(P3+tx)).x);
        (*(P+tx)).y = alpha_0 * ((*(P0+tx)).y) + alpha_1 * ((*(P1+tx)).y) + alpha_2 * ((*(P2+tx)).y) + alpha_3 * ((*(P3+tx)).y);
    }
}

void zmatrix_poly4_eval( unsigned long n, double2* P, double alpha_0, double2* P0, double alpha_1, double2* P1, double alpha_2, double2* P2, double alpha_3, double2* P3 )
{
    const unsigned long grids = ( n + 1023 ) >> 10;
    kernel_assert( ( zmatrix_poly4_eval_kernel<<<grids, 1024>>>( n, P, alpha_0, P0, alpha_1, P1, alpha_2, P2, alpha_3, P3 ) ) );
}

//
// P = P0 + alpha_1 P1 + alpha_2 P2 + alpha_3 P3
//
void __global__
zmatrix_poly4_eval_kernel(unsigned long n, double2* P, double2* P0, double alpha_1, double2* P1, double alpha_2, double2* P2, double alpha_3, double2* P3 )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;
    if ( tx < n )
    {
        (*(P+tx)).x =  ((*(P0+tx)).x) + alpha_1 * ((*(P1+tx)).x) + alpha_2 * ((*(P2+tx)).x) + alpha_3 * ((*(P3+tx)).x);
        (*(P+tx)).y =  ((*(P0+tx)).y) + alpha_1 * ((*(P1+tx)).y) + alpha_2 * ((*(P2+tx)).y) + alpha_3 * ((*(P3+tx)).y);
    }
}

void zmatrix_poly4_eval( unsigned long n, double2* P, double2* P0, double alpha_1, double2* P1, double alpha_2, double2* P2, double alpha_3, double2* P3 )
{
    const unsigned long grids = ( n + 1023 ) >> 10;
    kernel_assert( ( zmatrix_poly4_eval_kernel<<<grids, 1024>>>( n, P, P0, alpha_1, P1, alpha_2, P2, alpha_3, P3 ) ) );
}



void __global__
zmatrix_poly4_eval_kernel( unsigned long n, double2* P, double2* P0, double alpha_1_r, double alpha_1_i, double2* P1, double alpha_2_r, double alpha_2_i, double2* P2, double alpha_3_r, double alpha_3_i, double2* P3 )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;
    if ( tx < n )
    {
        (*(P+tx)) = hipCadd( hipCadd( (*(P0+tx)),
                                    hipCmul( make_hipDoubleComplex( alpha_1_r, alpha_1_i ), (*(P1+tx)) )
                                  ),
                            hipCadd( hipCmul( make_hipDoubleComplex( alpha_2_r, alpha_2_i ), (*(P2+tx)) ),
                                    hipCmul( make_hipDoubleComplex( alpha_3_r, alpha_3_i ), (*(P3+tx)) )
                                  )
                         );
    }
}

void zmatrix_poly4_eval( unsigned long n, double2* P, double2* P0, double alpha_1_r, double alpha_1_i, double2* P1, double alpha_2_r, double alpha_2_i, double2* P2, double alpha_3_r, double alpha_3_i, double2* P3 )
{
    const unsigned long grids = ( n + 1023 ) >> 10;
    kernel_assert( ( zmatrix_poly4_eval_kernel<<<grids, 1024>>>( n, P, P0, alpha_1_r, alpha_1_i, P1, alpha_2_r, alpha_2_i, P2, alpha_3_r, alpha_3_i, P3 ) ) );
}


/*
void __global__
zmatrix_poly4_eval_kernel(unsigned long n, double2* P, double2 alpha_0, double2* P0, double2 alpha_1, double2* P1, double2 alpha_2, double2* P2, double2 alpha_3, double2* P3 )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 256;
    if ( tx < n )
        (*(P+tx)) = cuAdd( cuAdd( cuMul( alpha_0, *(P0+tx) ), cuMul( alpha_1, *(P1+tx) ) ), cuAdd( cuMul( alpha_2, *(P2+tx) ), cuMul( alpha_3, *(P3+tx) ) ) );
}

void zmatrix_poly4_eval( unsigned long n, double2* P, double2 alpha_0, double2* P0, double2 alpha_1, double2* P1, double2 alpha_2, double2* P2, double2 alpha_3, double2* P3 )
{
    const unsigned long grids = ( n + 255 ) >> 8;
    kernel_assert( ( zmatrix_poly4_eval_kernel<<<grids, 256>>>( n, P, alpha_0, P0, alpha_1, P1, alpha_2, P2, alpha_3, P3 ) ) );
}
*/
