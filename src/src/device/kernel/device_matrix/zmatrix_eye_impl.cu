#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <f/device/assert/kernel_assert.hpp>

/*
void __global__
zmatrix_eye_kernel( double2* data, unsigned long const n )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;

    if ( tx < n )
        (*(data+tx*n+tx)).x = 1.0;
}

void zmatrix_eye_impl( double2* data, unsigned long const n )
{
    const unsigned long grids = ( n + 1023 ) >> 10;

    kernel_assert( (zmatrix_eye_kernel<<<grids, 1024>>>( data, n ) ) );
}
*/

void __global__
zmatrix_eye_kernel( double2* data, unsigned long const n, double val )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;

    if ( tx < n )
        (*(data+tx*n+tx)).x = val;
}

void zmatrix_eye_impl( double2* data, unsigned long const n, double val )
{
    const unsigned long grids = ( n + 1023 ) >> 10;

    kernel_assert( (zmatrix_eye_kernel<<<grids, 1024>>>( data, n, val ) ) );
}

void zmatrix_eye_impl( double2* data, unsigned long const n )
{
    zmatrix_eye_impl( data, n, 1.0 );
}

void __global__
zmatrix_eye_kernel( double2* data, unsigned long const n, double real, double imag )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;

    if ( tx < n )
    {
        (*(data+tx*n+tx)).x = real;
        (*(data+tx*n+tx)).x = imag;
    }
}

void zmatrix_eye_impl( double2* data, unsigned long const n, double real, double imag )
{
    const unsigned long grids = ( n + 1023 ) >> 10;

    kernel_assert( (zmatrix_eye_kernel<<<grids, 1024>>>( data, n, real, imag ) ) );
}

