#include "hip/hip_runtime.h"
#include <f/device/assert/cuda_assert.hpp>
#include <f/device/assert/kernel_assert.hpp>

#include <hip/hip_runtime.h>

void __global__ 
zmatrix_memcpy_real_imag_kernel( double2* device_data, double* device_real, double* device_imag, unsigned long length )
{
    const unsigned long tx = threadIdx.x + blockIdx.x * 1024;

    if ( tx < length )
    {
        (*(device_data+tx)).x = (*(device_real+tx));
        (*(device_data+tx)).y = (*(device_imag+tx));
    }
}

void zmatrix_memcpy_real_imag_impl( double2* device_data, const double* host_real, const double* host_imag, unsigned long length )
{
    double* device_real;
    double* device_imag;

    const unsigned long size_to_malloc = length * sizeof( double );
    cuda_assert( hipMalloc( (void**)&device_real, size_to_malloc ) );
    cuda_assert( hipMalloc( (void**)&device_imag, size_to_malloc ) );

    cuda_assert( hipMemcpy( device_real, host_real, size_to_malloc, hipMemcpyHostToDevice) );
    cuda_assert( hipMemcpy( device_imag, host_imag, size_to_malloc, hipMemcpyHostToDevice) );

    const unsigned long grids = ( length + 1023 ) >> 10;

    kernel_assert( ( zmatrix_memcpy_real_imag_kernel<<< grids, 1024>>>( device_data, device_real, device_imag, length ) ) );

    cuda_assert( hipFree( device_real ) );
    cuda_assert( hipFree( device_imag ) );
}

