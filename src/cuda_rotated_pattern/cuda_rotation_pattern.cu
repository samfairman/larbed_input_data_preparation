#include "hip/hip_runtime.h"
#include <f/device/device_assert/cuda_assert.hpp>
#include <f/device/device_assert/cublas_assert.hpp>
#include <f/device/device_assert/kernel_assert.hpp>

#include <hip/hip_runtime.h>

#include <hip/hip_complex.h>
#include <math_functions.h>
#include <cmath>
/*
__global__
void
impl_rotation_diag( double lambda, double kx_scaler, double kx_offset, double ky_scaler, double ky_offset, double rotation, double* tilt, double* gvec, double* cuda_diag, unsigned long const tilt_size, unsigned long max_dim, double cos_rot, double sin_rot )
{
    unsigned long const index = blockDim.x * blockIdx.x + threadIdx.x;
    if ( index >= tilt_size ) return;
    double const theta_x = tilt[index+index];
    double const theta_y = tilt[index+index+1];
    double const tx = (1.0+kx_scaler*10.0) * theta_x + kx_offset;
    double const ty = (1.0+ky_scaler*10.0) * theta_y + ky_offset;
    double const kx_ = sin(tx) / lambda;
    double const ky_ = sin(ty) / lambda;
    double const kx = kx_ * cos_rot + ky_ * sin_rot;
    double const ky = - kx_ * sin_rot + ky_ * cos_rot;

    double gx = 0.0;
    double gy = 0.0;

    for ( unsigned long jndex = 0; jndex < max_dim; ++jndex )
    {
       gx = gvec[jndex+jndex];
       gy = gvec[jndex+jndex+1];
       cuda_diag[jndex*tilt_size+index] = - gx*gx - gy*gy - 2.0 * ( kx*gx + ky*gy );
    }
}
*/
__global__
void
impl_rotation_diag( double lambda, double kx_scaler, double kx_offset, double ky_scaler, double ky_offset, double rotation, double* tilt, double* gvec, double* cuda_diag, unsigned long const tilt_size, unsigned long max_dim, double cos_rot, double sin_rot )
{
    unsigned long const index = blockDim.x * blockIdx.x + threadIdx.x;
    if ( index >= tilt_size ) return;
    double const theta_x = tilt[index+index];
    double const theta_y = tilt[index+index+1];
    double const tx = (1.0+kx_scaler*10.0) * theta_x + kx_offset;
    double const ty = (1.0+ky_scaler*10.0) * theta_y + ky_offset;
    double const kx_ = sin(tx) / lambda;
    double const ky_ = sin(ty) / lambda;
    double const kx = kx_ * cos_rot + ky_ * sin_rot;
    double const ky = - kx_ * sin_rot + ky_ * cos_rot;

    double gx = 0.0;
    double gy = 0.0;

    for ( unsigned long jndex = 0; jndex < max_dim; ++jndex )
    {
       gx = gvec[jndex+jndex];
       gy = gvec[jndex+jndex+1];
       //cuda_diag[jndex*tilt_size+index] = - gx*gx - gy*gy - 2.0 * ( kx*gx + ky*gy );
       cuda_diag[index*max_dim+jndex] = - gx*gx - gy*gy - 2.0 * ( kx*gx + ky*gy );
    }
}

void make_rotation_diag( double lambda, double kx_scaler, double kx_offset, double ky_scaler, double ky_offset, double rotation, double* tilt, double* gvec, double* cuda_diag, unsigned long const tilt_size, unsigned long max_dim )
{
    unsigned long const threads = 64;
    unsigned long const grids = (tilt_size+threads-1)/threads;

	rotation = -1.88 * (1.0+rotation*10.0);
    double const cos_rot = std::cos( rotation );
    double const sin_rot = std::sin( rotation );

    kernel_assert( (impl_rotation_diag<<<grids,threads>>>( lambda, kx_scaler, kx_offset, ky_scaler, ky_offset, rotation, tilt, gvec, cuda_diag, tilt_size, max_dim, cos_rot, sin_rot )) );
    cuda_assert( hipDeviceSynchronize() );
}

